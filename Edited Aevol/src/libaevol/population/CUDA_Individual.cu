#include "hip/hip_runtime.h"
#include "CUDA_Individual.h"
#include "CUDA_Individual_cu.h"
#include <cstdint>
#include <stdio.h>
#include <unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

#include "ExpManager.h"
#include "HybridFuzzy.h"

#define DEBUG 1
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n",
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
  if (err != hipSuccess)
  {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
            file, line );
    exit( EXIT_FAILURE );
  }
}

void cuda_init() {
  size_t limit = 9000000000;
  hipDeviceSetLimit(hipLimitMallocHeapSize, limit);
  hipDeviceSetLimit(hipLimitStackSize, 1024*1024);
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 64*1024*1024);
}

void transfer_in(ExpManager* exp_m, bool init_all_struct) {
  if (init_all_struct) {
    hipDeviceReset();
    hipDeviceSynchronize();
  }

  size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  hipError_t result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
  if( result == hipSuccess )
  {
    printf( "Total Memory: %d MB, Free Memory: %d MB\n",
            uTotalMemoryInBytes / ( 1024 * 1024 ),
            uCurAvailMemoryInBytes / ( 1024 * 1024 ));
  }

  host_dna = (char**)malloc(exp_m->nb_indivs()*sizeof(char*));
  checkCuda(hipMalloc((void***)&dna,exp_m->nb_indivs()*sizeof(char*)));

  //host_dna_lead_promoter = (int8_t**)malloc(exp_m->nb_indivs()*sizeof(int8_t*));
  //checkCuda(hipMalloc((void***)&dna_lead_promoter,exp_m->nb_indivs()*sizeof(int8_t*)));
  //host_dna_lag_promoter = (int8_t**)malloc(exp_m->nb_indivs()*sizeof(int8_t*));
  //checkCuda(hipMalloc((void***)&dna_lag_promoter,exp_m->nb_indivs()*sizeof(int8_t*)));

  host_dna_lead_term = (int8_t**)malloc(exp_m->nb_indivs()*sizeof(int8_t*));
  checkCuda(hipMalloc((void***)&dna_lead_term,exp_m->nb_indivs()*sizeof(int8_t*)));
  host_dna_lag_term = (int8_t**)malloc(exp_m->nb_indivs()*sizeof(int8_t*));
  checkCuda(hipMalloc((void***)&dna_lag_term,exp_m->nb_indivs()*sizeof(int8_t*)));

  host_phenotype = (float**)malloc(exp_m->nb_indivs()*sizeof(float*));
  checkCuda(hipMalloc((void***)&phenotype,exp_m->nb_indivs()*sizeof(float*)));

  checkCuda(hipMalloc((void**)&dna_size,
             exp_m->nb_indivs() * sizeof(size_t)));
  size_t* host_dna_size = (size_t*)malloc(exp_m->nb_indivs()*sizeof(size_t));


  checkCuda(hipMalloc((void**)&nb_promoters,
             exp_m->nb_indivs() * sizeof(int)));
  checkCuda(hipMemset(nb_promoters, 0, exp_m->nb_indivs() * sizeof(int)));

  if (init_all_struct) {
    printf("Init struct");
    checkCuda(hipMalloc((void**) &max_nb_elements_rna_list,
                         exp_m->nb_indivs() * sizeof(int)));
    checkCuda(hipMemset(max_nb_elements_rna_list, 0, exp_m->nb_indivs() * sizeof(int)));

    checkCuda(hipMalloc((void**) &max_nb_elements_protein_list,
                         exp_m->nb_indivs() * sizeof(int)));
    checkCuda(hipMemset(max_nb_elements_protein_list, 0, exp_m->nb_indivs() * sizeof(int)));
  }

  checkCuda(hipMalloc((void**)&metaerror,
             exp_m->nb_indivs() * sizeof(float)));


  checkCuda(hipMalloc((void**)&fitness,
             exp_m->nb_indivs() * sizeof(double)));


  checkCuda(hipMalloc((void**)&nb_protein,
             exp_m->nb_indivs() * sizeof(int32_t)));
  checkCuda(hipMemset(nb_protein, 0, exp_m->nb_indivs() * sizeof(int32_t)));


  checkCuda(hipMalloc((void**)&max_nb_rna,
                       sizeof(int32_t)));
  checkCuda(hipMemset(max_nb_rna, 0, sizeof(int32_t)));


  checkCuda(hipMalloc((void**)&max_nb_protein,
                       sizeof(int32_t)));
  checkCuda(hipMemset(max_nb_protein, 0, sizeof(int32_t)));

  checkCuda(hipMalloc((void**)&idx_rna,
             exp_m->nb_indivs() * sizeof(int32_t)));
  checkCuda(hipMemset(idx_rna, 0, exp_m->nb_indivs() * sizeof(int32_t)));

  if (init_all_struct) checkCuda(hipMalloc((void***)&protein_list,exp_m->nb_indivs()*sizeof(cProtein*)));

  checkCuda(hipMalloc((void**)&idx_protein,
                       exp_m->nb_indivs() * sizeof(int32_t)));
  checkCuda(hipMemset(idx_protein, 0, exp_m->nb_indivs() * sizeof(int32_t)));

  if (init_all_struct) checkCuda(hipMalloc((void***)&rna,exp_m->nb_indivs()*sizeof(cRNA**)));

  checkCuda(hipMalloc((void***)&dynPromoterList,
                       exp_m->nb_indivs()*sizeof(pStruct*)));
  host_dynPromoterList = (pStruct**)malloc(exp_m->nb_indivs()*sizeof(pStruct*));

  checkCuda(hipMalloc((void***)&dynTerminatorList,
                       exp_m->nb_indivs()*sizeof(pStruct*)));
  host_dynTerminatorList = (pStruct**)malloc(exp_m->nb_indivs()*sizeof(pStruct*));

  result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
  if( result == hipSuccess )
  {
    printf( "Total Memory: %d MB, Free Memory: %d MB\n",
            uTotalMemoryInBytes / ( 1024 * 1024 ),
            uCurAvailMemoryInBytes / ( 1024 * 1024 ));
  }


  int x,y,max_dna=-1;

  for (int i = 0; i < exp_m->nb_indivs(); i++) {

    x = i / exp_m->grid_height();
    y = i % exp_m->grid_height();

    checkCuda(hipMalloc((void**) &host_dna[i],
               exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(char)));

    checkCuda(hipMemcpy(host_dna[i], exp_m->world()->indiv_at(x, y)->genetic_unit(0).dna()->data(),
               exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(char),
               hipMemcpyHostToDevice));

    host_dna_size[i] = exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length();
    max_dna = max_dna < exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() ?
              exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() : max_dna;

    //checkCuda(hipMalloc((void**) &host_dna_lead_promoter[i],
    //           exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(int8_t)));

    //checkCuda(hipMalloc((void**) &host_dna_lag_promoter[i],
    //           exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(int8_t)));

    checkCuda(hipMalloc((void**) &host_dna_lead_term[i],
               exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(int8_t)));

    checkCuda(hipMalloc((void**) &host_dna_lag_term[i],
               exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(int8_t)));


    checkCuda(hipMalloc((void**) &host_phenotype[i], 300 * sizeof(float)));
    checkCuda(hipMemset(host_phenotype[i], 0.0, 300 * sizeof(float)));

    checkCuda(hipMalloc((void**) &host_dynPromoterList[i], PROMOTER_ARRAY_SIZE * sizeof(pStruct)));
    checkCuda(hipMalloc((void**) &host_dynTerminatorList[i], exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length() * sizeof(pStruct)));
  }

  host_max_dna_size= max_dna;

  checkCuda(hipMemcpy(dna,host_dna,exp_m->nb_indivs()*sizeof(char*),hipMemcpyHostToDevice));

  //checkCuda(hipMemcpy(dna_lag_promoter,host_dna_lag_promoter,exp_m->nb_indivs()*sizeof(int8_t*),hipMemcpyHostToDevice));
  //checkCuda(hipMemcpy(dna_lead_promoter,host_dna_lead_promoter,exp_m->nb_indivs()*sizeof(int8_t*),hipMemcpyHostToDevice));


  checkCuda(hipMemcpy(phenotype,host_phenotype,exp_m->nb_indivs()*sizeof(float*),hipMemcpyHostToDevice));

  checkCuda(hipMemcpy(dna_lag_term,host_dna_lag_term,exp_m->nb_indivs()*sizeof(int8_t*),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(dna_lead_term,host_dna_lead_term,exp_m->nb_indivs()*sizeof(int8_t*),hipMemcpyHostToDevice));

  checkCuda(hipMemcpy(dynPromoterList,host_dynPromoterList,exp_m->nb_indivs()*sizeof(pStruct*),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(dynTerminatorList,host_dynTerminatorList,exp_m->nb_indivs()*sizeof(pStruct*),hipMemcpyHostToDevice));

  checkCuda(hipMemcpy(dna_size,
             host_dna_size, exp_m->nb_indivs() * sizeof(size_t), hipMemcpyHostToDevice));

  /*free(host_dna_size);
  free(host_phenotype);
  free(host_dna_lag_term);
  free(host_dna_lead_term);
  free(host_dna_lag_promoter);
  free(host_dna_lead_promoter);
  free(host_dna);*/

  checkCuda(hipMalloc((void**) &target,
             300 * sizeof(float)));

  float target_host[300];
  for (int i = 0; i < 300; i++) {
    target_host[i] = ((HybridFuzzy*)exp_m->world()->phenotypic_target_handler()->phenotypic_target().fuzzy())->points()[i];
  }

  checkCuda(hipMemcpy(target,
                       target_host,
             300 * sizeof(float), hipMemcpyHostToDevice));

}

void print_debug_promoters_start(ExpManager* exp_m) {
  print_debug_promoters_start(exp_m,737);
}

void print_debug_promoters_start(ExpManager* exp_m, int i) {
  int x,y;

  //for (int i = 0; i < exp_m->nb_indivs(); i++) {
    x = i / exp_m->grid_height();
    y = i % exp_m->grid_height();

    for (auto& strand_id: {LEADING, LAGGING}) {
      if (strand_id == LEADING) printf("Individual %d (CPU) Promoters : LEADING ",i);
      else printf("Individual %d (CPU) Promoters : LAGGING ",i);
      auto& strand = exp_m->world()->indiv_at(x, y)->genetic_unit(0).rna_list()[strand_id];
      for (auto rna = strand.begin(); rna != strand.end(); ++rna) {
        printf("%d ",rna->promoter_pos());
      }
      printf("\n");
    }

    debug_promoter_start<<<1,1>>>(dna_size,dynPromoterList,nb_promoters,
        i);
printf("Terminator !!! \n");
  debug_promoter_stop<<<1,1>>>(dna_size,dna_lead_term,dna_lag_term,nb_promoters,
      i);
  hipDeviceSynchronize();
  //}
  printf("Terminator END !!! \n");
}

void print_debug_rna(ExpManager* exp_m) {
  print_debug_rna(exp_m,737);
}

void print_debug_rna(ExpManager* exp_m, int i) {
  int x,y;
  //for (int i = 0; i < exp_m->nb_indivs(); i++) {
    x = i / exp_m->grid_height();
    y = i % exp_m->grid_height();

  int prot_idx = 0;

int rna_idx = 0;
    for (auto& strand_id: {LEADING, LAGGING}) {
      printf("%d -- Individual %d (%d %d CPU) \n",AeTime::time(),i,x,y);
      auto& strand = exp_m->world()->indiv_at(x, y)->genetic_unit(0).rna_list()[strand_id];
      for (auto rna = strand.begin(); rna != strand.end(); ++rna) {
        printf("RNA %d : ",rna_idx);

        if (strand_id == LEADING) printf("LEADING ");
        else printf("LAGGING ");

        printf("%d (%d) %d %f (size %d)\n",rna->promoter_pos(),
               rna->first_transcribed_pos(),
            rna->last_transcribed_pos(),
            rna->basal_level(),rna->transcript_length());

/*
        for (auto prot : rna->transcribed_proteins()) {
          printf("Protein CPU (%d) %d : %d %d (%d %d) %f %f %f\n",prot_idx,
                 rna_idx,
                 prot->shine_dal_pos(),
                 prot->last_STOP_base_pos(),
                 prot->first_translated_pos(),
                 prot->last_translated_pos(),
                 prot->mean(),
                 prot->height(),
                 prot->width());
          prot_idx++;
        }*/
      rna_idx++;
      }

    }

    debug_rna<<<1,1>>>(dna_size,dna_lead_promoter,dna_lag_promoter,rna,idx_rna,
        i);
  //}

}

void print_debug_protein(ExpManager* exp_m) {
  print_debug_protein(exp_m,737);
}

void print_debug_protein(ExpManager* exp_m, int i) {
  int x,y;

  //for (int i = 0; i < exp_m->nb_indivs(); i++) {
  x = i / exp_m->grid_height();
  y = i % exp_m->grid_height();

  int prot_idx = 0;
  printf("Protein list size on CPU %d (%d %d): %d (%lu)\n",i,x,y,exp_m->world()->indiv_at(x, y)->protein_list().size(),
         exp_m->world()->indiv_at(x, y)->genetic_unit(0).seq_length());
  for (auto prot : exp_m->world()->indiv_at(x, y)->protein_list()) {
    printf("Protein CPU (%d) %d : %d %d (%d %d) %lf %lf %lf isfunctional %d\n",prot_idx,
           prot->rna_list().size(),
           prot->shine_dal_pos(),
           prot->last_STOP_base_pos(),
           prot->first_translated_pos(),
           prot->last_translated_pos(),
           prot->mean(),
           prot->height(),
           prot->width(),
           prot->is_functional());
   /* int cod_idx = 0;
    for (auto cod : prot->AA_list()) {
      printf("COD[%d] = %d\n",cod_idx,cod->value());
      cod_idx++;
    }*/
  }
  debug_protein<<<1,1>>>(idx_protein,protein_list,dna,
      i);
  //}

}

void print_debug_phenotype(ExpManager* exp_m) {
  print_debug_phenotype(exp_m,737);
}

void print_debug_phenotype(ExpManager* exp_m, int i) {
  int x,y;
  //for (int i = 0; i < exp_m->nb_indivs(); i++) {
  x = i / exp_m->grid_height();
  y = i % exp_m->grid_height();

  exp_m->world()->indiv_at(x, y)->phenotype()->print();

  debug_phenotype<<<1,1>>>(phenotype,target,metaerror,fitness,
      i);
}

void print_debug_fitness(ExpManager* exp_m) {
  int x,y;
  int i = 15;
  //for (int i = 0; i < exp_m->nb_indivs(); i++) {
  x = i / exp_m->grid_height();
  y = i % exp_m->grid_height();

  //printf("%d %d\n",x,y);

  /*debug_fitness<<<1,1>>>(phenotype, target,
      metaerror, fitness,
      i);*/
}

void transfer_out(ExpManager* exp_m, bool delete_all_struct) {
  printf("Starting transfert\n");

  float* host_metaerror = (float*)malloc(exp_m->nb_indivs()*sizeof(float));
  double* host_fitness = (double*)malloc(exp_m->nb_indivs()*sizeof(double));
  /*size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  hipError_t result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
  if( result == hipSuccess )
  {
    printf( "Total Memory: %d MB, Free Memory: %d MB\n",
            uTotalMemoryInBytes / ( 1024 * 1024 ),
            uCurAvailMemoryInBytes / ( 1024 * 1024 ));
  }*/
  hipDeviceSynchronize();
  //printf("Transfering Metaerror\n");
  checkCuda(hipMemcpy(host_metaerror,
             metaerror, exp_m->nb_indivs() * sizeof(float), hipMemcpyDeviceToHost));
  //printf("Transfering Fitness\n");
  checkCuda(hipMemcpy(host_fitness,
             fitness, exp_m->nb_indivs() * sizeof(double), hipMemcpyDeviceToHost));
  printf("Transfer END\n");

  bool error_detected=false;

  int x,y;
  for (int i = 0; i < exp_m->nb_indivs(); i++) {
    x = i / exp_m->grid_height();
    y = i % exp_m->grid_height();

    double fit_1 = exp_m->world()->indiv_at(x, y)->dist_to_target_by_feature(METABOLISM);
    double fit_2 = host_metaerror[i];
    float i_fit_1 = roundf(fit_1*100);
    float i_fit_2 = roundf(fit_2*100);


    if (i_fit_1 != i_fit_2) {
      printf(
          "ERROR -- Individual %d : Metaerror (CPU/GPU) : %e -- %e || Fitness (CPU/GPU) : %e -- %e\n",
          i,
          exp_m->world()->indiv_at(x, y)->dist_to_target_by_feature(METABOLISM),
          host_metaerror[i],
          exp_m->world()->indiv_at(x, y)->fitness(), host_fitness[i]);


      /*if (i == 0) {
        print_debug_promoters_start(exp_m,i);
        print_debug_rna(exp_m,i);
        print_debug_protein(exp_m,i);
      }*/

      //print_debug_promoters_start(exp_m,i);
      //print_debug_rna(exp_m,i);
      //print_debug_protein(exp_m,i);
      //print_debug_phenotype(exp_m,i);

      //char c=getchar();
      //printf("Read %c\n",c);
      //if (c=='q') { error_detected = true;}
    }
  }

  //free_list<<<1024,1>>>(protein_list,rna,idx_protein,idx_rna);

 for (int i = 0; i < exp_m->nb_indivs(); i++) {
    HANDLE_ERROR(hipFree(host_dna[i]));
   //HANDLE_ERROR(hipFree(host_dna_lead_promoter[i]));
   //HANDLE_ERROR(hipFree(host_dna_lag_promoter[i]));
   HANDLE_ERROR(hipFree(host_dna_lead_term[i]));
   HANDLE_ERROR(hipFree(host_dna_lag_term[i]));
   HANDLE_ERROR(hipFree(host_phenotype[i]));
   HANDLE_ERROR(hipFree(host_dynPromoterList[i]));
   HANDLE_ERROR(hipFree(host_dynTerminatorList[i]));
  }

  HANDLE_ERROR(hipFree(nb_promoters));
  if (delete_all_struct) HANDLE_ERROR(hipFree(max_nb_elements_rna_list));

  HANDLE_ERROR(hipFree(nb_protein));
  HANDLE_ERROR(hipFree(metaerror));
  HANDLE_ERROR(hipFree(fitness));
  HANDLE_ERROR(hipFree(idx_protein));
  HANDLE_ERROR(hipFree(idx_rna));
  if (delete_all_struct) HANDLE_ERROR(hipFree(protein_list));
  if (delete_all_struct) HANDLE_ERROR(hipFree(rna));
  HANDLE_ERROR(hipFree(target));

  HANDLE_ERROR(hipFree(dna));
  //HANDLE_ERROR(hipFree(dna_lead_promoter));
  //HANDLE_ERROR(hipFree(dna_lag_promoter));
  HANDLE_ERROR(hipFree(dna_lead_term));
  HANDLE_ERROR(hipFree(dna_lag_term));
  HANDLE_ERROR(hipFree(phenotype));
  HANDLE_ERROR(hipFree(dynPromoterList));
  HANDLE_ERROR(hipFree(dynTerminatorList));

  /*if (error_detected)
    exit(-42);*/
}

void run_a_step(int nb_indiv,float w_max, double selection_pressure, bool first_gen) {
  cuda_init();
  nb_indiv = 1024;


  /*limit=0;
  hipDeviceGetLimit(&limit, hipLimitStackSize);
  printf("hipLimitStackSize: %u\n", (unsigned)limit);
  hipDeviceGetLimit(&limit, hipLimitPrintfFifoSize);
  printf("hipLimitPrintfFifoSize: %u\n", (unsigned)limit);
  hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
  printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);

  size_t uCurAvailMemoryInBytes;
  size_t uTotalMemoryInBytes;
  hipError_t result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
  if( result == hipSuccess )
  {
    printf( "Total Memory: %d MB, Free Memory: %d MB\n",
            uTotalMemoryInBytes / ( 1024 * 1024 ),
            uCurAvailMemoryInBytes / ( 1024 * 1024 ));
  }*/

  printf("Nb individual %d / max DNA size %d\n",nb_indiv,host_max_dna_size);



  int block_size = 1 + host_max_dna_size / 65000;
  int y_dim_size = host_max_dna_size / block_size;
  int x_dim_size = 1024 * block_size;

  //int bucket_size = 1;
  int tmp_block = 1 + (( y_dim_size * 52 ) / 1024);

  int bucket_size = 19;
  int thread_number = bucket_size * 52;

  y_dim_size = 1 + y_dim_size / 19;

  dim3 dimGrid(x_dim_size,y_dim_size);

  //printf("Dim grid %d %d\n",x_dim_size,y_dim_size);
  //hipDeviceSynchronize();
 // debug_dna<<<1,1>>>(dna_size, dna);
//  hipDeviceSynchronize();
  //init_array<<<1024,1>>>(nb_promoters);
  //search_start_RNA<<<dimGrid,44>>>(dna_size,dna,dna_lead_promoter,dna_lag_promoter,nb_promoters,dynPromoterList,block_size);

  //debug_promoter_start<<<1,1>>>(dna_size,dna_lead_promoter,dna_lag_promoter,nb_promoters,
  //    1);
  //return;
  //hipDeviceSynchronize();
  //search_stop_RNA<<<dimGrid,8>>>(dna_size,dna,dna_lead_term,dna_lag_term,block_size);
  //hipDeviceSynchronize();
  /*result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
  if( result == hipSuccess )
  {
    printf( "Total Memory: %d MB, Free Memory: %d MB\n",
            uTotalMemoryInBytes / ( 1024 * 1024 ),
            uCurAvailMemoryInBytes / ( 1024 * 1024 ));
  }*/

  //debug_promoter_stop<<<1,1>>>(dna_size,dna_lead_promoter,dna_lag_promoter,nb_promoters,
  //    1);


  printf("X Dim %d Y Dim %d Thread %d Bucket %d Block %d\n",x_dim_size,y_dim_size,thread_number,bucket_size,block_size);

  search_start_stop_RNA_bucket<<<dimGrid,thread_number>>>(dna_size,dna,dna_lead_promoter,dna_lag_promoter,
      nb_promoters,dynPromoterList,dna_lead_term,dna_lag_term,bucket_size,block_size);

  //hipDeviceSynchronize();

  init_RNA_struct<<<nb_indiv,1>>>(nb_indiv,rna,nb_promoters,max_nb_rna,idx_rna,max_nb_elements_rna_list);
  //hipDeviceSynchronize();

  int max_promoters_host;
  HANDLE_ERROR(hipMemcpy(&max_promoters_host,
                          max_nb_rna, sizeof(int32_t), hipMemcpyDeviceToHost));

  //display_size_dna<<<1024,1>>>(dna_size);

  int threads_size = 1 + max_promoters_host / 1024;
  y_dim_size = max_promoters_host / threads_size;
  x_dim_size = 1024 * threads_size;

  printf("Max RNA %d (%d %d)\n",max_promoters_host,x_dim_size,y_dim_size);

  compute_RNA<<<x_dim_size,y_dim_size>>>(nb_indiv,dna_lead_promoter,dna_lag_promoter,dna_lead_term,dna_lag_term,
                            dna,dna_size,rna,idx_rna,nb_promoters,dynPromoterList,threads_size,y_dim_size);


  //hipDeviceSynchronize();
  //debug_rna<<<1,1>>>(dna_size,dna_lead_promoter,dna_lag_promoter,rna,idx_rna,
  //    1020);

  //printf("Max RNA is computed\n");
  max_rna<<<nb_indiv,1>>>(idx_rna,max_nb_rna);
  //hipDeviceSynchronize();
//  printf("Max RNA is transfering\n");

  int max_rna_host;
  HANDLE_ERROR(hipMemcpy(&max_rna_host,
             max_nb_rna, sizeof(int32_t), hipMemcpyDeviceToHost));

  //display_size_dna<<<1024,1>>>(dna_size);

  threads_size = 1 + max_rna_host / 1024;
  y_dim_size = max_rna_host / threads_size;
  x_dim_size = 1024 * threads_size;

  printf("Max RNA %d (%d %d)\n",max_rna_host,x_dim_size,y_dim_size);

  compute_start_protein<<<x_dim_size,y_dim_size>>>(idx_rna,rna,dna,dna_size,nb_protein,
      threads_size,y_dim_size);
  //hipDeviceSynchronize();

/*
  result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );
  if( result == hipSuccess )
  {
    printf( "Total Memory: %d MB, Free Memory: %d MB\n",
            uTotalMemoryInBytes / ( 1024 * 1024 ),
            uCurAvailMemoryInBytes / ( 1024 * 1024 ));
  }
*/
  init_protein_struct<<<x_dim_size,y_dim_size>>>(nb_indiv,nb_protein,protein_list,
      idx_protein,rna,idx_rna,max_nb_protein,max_nb_elements_protein_list,threads_size,y_dim_size);
  //hipDeviceSynchronize();

  int max_nb_protein_host;
  HANDLE_ERROR(hipMemcpy(&max_nb_protein_host,
             max_nb_protein, sizeof(int32_t), hipMemcpyDeviceToHost));

  threads_size = 1 + max_nb_protein_host / 1024;
  y_dim_size = max_nb_protein_host / threads_size;
  x_dim_size = max_rna_host * threads_size;

  block_size = 1 + x_dim_size / 65000;
  x_dim_size = x_dim_size / block_size;
  int z_dim_size = 1024 * block_size;

  dim3 dimGrid2(z_dim_size,x_dim_size);

  printf("Max Protein %d (%d %d %d)\n",max_nb_protein_host,z_dim_size,x_dim_size,y_dim_size);

  //display_size_dna<<<1024,1>>>(dna_size);
  //hipDeviceSynchronize();

  compute_protein<<<dimGrid2,y_dim_size>>>(rna,protein_list,idx_protein,dna_size,dna,idx_rna, threads_size,y_dim_size,block_size);
  //hipDeviceSynchronize();

  max_protein<<<nb_indiv,1>>>(max_nb_protein,idx_protein);
  HANDLE_ERROR(hipMemcpy(&max_nb_protein_host,
             max_nb_protein, sizeof(int32_t), hipMemcpyDeviceToHost));


  threads_size = 1 + max_nb_protein_host / 1024;
  y_dim_size = max_nb_protein_host / threads_size;
  x_dim_size = 1024 * threads_size;

  printf("Max Protein Updated %d (%d %d)\n",max_nb_protein_host,x_dim_size,y_dim_size);

  translate_protein<<<x_dim_size,y_dim_size>>>(w_max,idx_protein,protein_list,dna,dna_size,threads_size,y_dim_size);
  compute_phenotype<<<x_dim_size,y_dim_size>>>(idx_protein,protein_list,phenotype,threads_size,y_dim_size);
  compute_metaerror_fitness<<<nb_indiv,300>>>(selection_pressure,phenotype,target,metaerror,fitness);
  hipDeviceSynchronize();

  /*if (AeTime::time()==14) {
    hipDeviceSynchronize();
    hipProfilerStop();
    exit(-1);
  }*/

}

__global__
void init_array(int* nb_promoters) {
  int indiv_id = blockIdx.x;

  nb_promoters[indiv_id] = 0;
  //atomicAdd(nb_promoters+ indiv_id,1);
  //atomicAdd(&nb_promoters[indiv_id],1);
  //printf("%d : %d\n",indiv_id,nb_promoters[indiv_id]);
}

__global__
void search_start_RNA(size_t* dna_size, char** dna, int8_t** dna_lead_promoter,
                      int8_t** dna_lag_promoter, int* nb_promoters,
                      pStruct** dynPromoterList, int block_size) {
  int indiv_id = blockIdx.x / block_size;
  int block_id = blockIdx.x % block_size;
  int pos_block_size = blockIdx.y;

  int dna_pos = gridDim.y*block_id+pos_block_size;

    __shared__ int dist_leading[22];
    __shared__ int dist_lagging[22];

    int motif_id = threadIdx.x;
    bool leading_or_lagging = true;

    if (dna_size[indiv_id] < PROM_SIZE) {
      //printf("START -- SMALL SIZE\n");
      /*if (dna_pos < dna_size[indiv_id] && threadIdx.x == 0) {
        dna_lead_promoter[indiv_id][dna_pos] = 0;
        dna_lag_promoter[indiv_id][dna_pos] = 0;
      }*/
    } else if (dna_pos < dna_size[indiv_id]) {
      if (motif_id >= 22) {
        // LAGGING
        motif_id -= 22;

        int pos = dna_pos - motif_id < 0 ?
                  dna_size[indiv_id] + (dna_pos - motif_id) :
                  dna_pos - motif_id;


        char s_motif=PROM_SEQ_LAG[motif_id];

        /*if (pos < 0 || pos > dna_size[indiv_id])
          printf("Checking DNA at %d (dna pos %d motif %d length %lu) for indiv %d (block %d)\n",
                 pos,dna_pos,motif_id,dna_size[indiv_id],indiv_id,block_id);*/

        char s_dna = dna[indiv_id][pos];

        dist_lagging[motif_id] =
            s_motif == s_dna ? 0 : 1;

        leading_or_lagging = true;
      } else {
        // LEADING
        int pos = dna_pos + motif_id >= dna_size[indiv_id] ?
                  dna_pos + motif_id - dna_size[indiv_id] : dna_pos + motif_id;

        dist_leading[motif_id] =
            PROM_SEQ_LEAD[motif_id] == dna[indiv_id][pos] ? 0 : 1;

        leading_or_lagging = false;
      }

      __syncthreads();

      if (threadIdx.x == 0) {
        int dist_lead = dist_leading[0] + dist_leading[1] + dist_leading[2] +
                        dist_leading[3] +
                        dist_leading[4] + dist_leading[5] + dist_leading[6] +
                        dist_leading[7] + dist_leading[8] +
                        dist_leading[9] + dist_leading[10] + dist_leading[11] +
                        dist_leading[12] + dist_leading[13] +
                        dist_leading[14] + dist_leading[15] + dist_leading[16] +
                        dist_leading[17] + dist_leading[18] +
                        dist_leading[19] + dist_leading[20] + dist_leading[21];

        int dist_lag = dist_lagging[0] + dist_lagging[1] + dist_lagging[2] +
                       dist_lagging[3] +
                       dist_lagging[4] + dist_lagging[5] + dist_lagging[6] +
                       dist_lagging[7] + dist_lagging[8] +
                       dist_lagging[9] + dist_lagging[10] + dist_lagging[11] +
                       dist_lagging[12] + dist_lagging[13] +
                       dist_lagging[14] + dist_lagging[15] + dist_lagging[16] +
                       dist_lagging[17] + dist_lagging[18] +
                       dist_lagging[19] + dist_lagging[20] + dist_lagging[21];


        //dna_lead_promoter[indiv_id][dna_pos] = dist_lead > 4 ? -1 : dist_lead;
 //       int nb_pro = dist_lead <= 4 ? 1 : 0;

        if (dist_lead <= 4) {
          int rna_idx = atomicAdd(nb_promoters + indiv_id, 1);

          dynPromoterList[indiv_id][rna_idx].pos = dna_pos;
          dynPromoterList[indiv_id][rna_idx].leading_or_lagging = true;
          dynPromoterList[indiv_id][rna_idx].error = dist_lead;
        }

        //dna_lag_promoter[indiv_id][dna_pos] = dist_lag > 4 ? -1 : dist_lag;
//        nb_pro += dist_lag <= 4 ? 1 : 0;

        //    if (indiv_id == 410 && dna_pos == 10)
        //      printf("Promoter found at %d ; %d %d \n",dna_pos,dist_lead,dist_lag);
        if (dist_lag <= 4) {
          int rna_idx = atomicAdd(nb_promoters + indiv_id, 1);

          dynPromoterList[indiv_id][rna_idx].pos = dna_pos;
          dynPromoterList[indiv_id][rna_idx].leading_or_lagging = false;
          dynPromoterList[indiv_id][rna_idx].error = dist_lag;
        }

      }

  }
}


__global__
void search_start_RNA_bucket(size_t* dna_size, char** dna, int8_t** dna_lead_promoter,
                      int8_t** dna_lag_promoter, int* nb_promoters,
                      pStruct** dynPromoterList, int bucket_size, int block_size) {
  int indiv_id = blockIdx.x / block_size;
  int block_id = blockIdx.x % block_size;
  int pos_block_size = blockIdx.y;

  int dna_pos = gridDim.y*block_id+pos_block_size;

  __shared__ int dist_leading[BUCKET_MAX_SIZE][22];
  __shared__ int dist_lagging[BUCKET_MAX_SIZE][22];

  int motif_id = threadIdx.x % 22;
  int dna_global_offset = threadIdx.x / 22;
  dna_pos+=dna_global_offset;

  if (dna_pos < dna_size[indiv_id] && dna_size[indiv_id] >= PROM_SIZE) {
      if (motif_id >= 22) {
        // LAGGING
        int t_motif_id = motif_id - 22;
        dist_lagging[dna_global_offset][t_motif_id] =
            PROM_SEQ_LAG[t_motif_id] == dna[indiv_id][dna_pos - t_motif_id < 0 ?
                                                    dna_size[indiv_id] + (dna_pos - t_motif_id) :
                                                    dna_pos - t_motif_id] ? 0 : 1;
      } else {
        // LEADING
        dist_leading[dna_global_offset][motif_id] =
            PROM_SEQ_LEAD[motif_id] == dna[indiv_id][dna_pos + motif_id >= dna_size[indiv_id] ?
                                                     dna_pos + motif_id - dna_size[indiv_id] : dna_pos + motif_id] ? 0 : 1;
      }


    __syncthreads();

    if (motif_id == 0) {

      int dist_lead = dist_leading[dna_global_offset][0] +
                      dist_leading[dna_global_offset][1] +
                      dist_leading[dna_global_offset][2] +
                      dist_leading[dna_global_offset][3] +
                      dist_leading[dna_global_offset][4] +
                      dist_leading[dna_global_offset][5] +
                      dist_leading[dna_global_offset][6] +
                      dist_leading[dna_global_offset][7] +
                      dist_leading[dna_global_offset][8] +
                      dist_leading[dna_global_offset][9] +
                      dist_leading[dna_global_offset][10] +
                      dist_leading[dna_global_offset][11] +
                      dist_leading[dna_global_offset][12] +
                      dist_leading[dna_global_offset][13] +
                      dist_leading[dna_global_offset][14] +
                      dist_leading[dna_global_offset][15] +
                      dist_leading[dna_global_offset][16] +
                      dist_leading[dna_global_offset][17] +
                      dist_leading[dna_global_offset][18] +
                      dist_leading[dna_global_offset][19] +
                      dist_leading[dna_global_offset][20] +
                      dist_leading[dna_global_offset][21];

      if (dist_lead <= 4) {
        int rna_idx = atomicAdd(nb_promoters + indiv_id, 1);

        dynPromoterList[indiv_id][rna_idx].pos = dna_pos;
        dynPromoterList[indiv_id][rna_idx].leading_or_lagging = true;
        dynPromoterList[indiv_id][rna_idx].error = dist_lead;
      }
    }

    if (motif_id == 22) {
      int dist_lag = dist_lagging[dna_global_offset][0] +
                     dist_lagging[dna_global_offset][1] +
                     dist_lagging[dna_global_offset][2] +
                     dist_lagging[dna_global_offset][3] +
                     dist_lagging[dna_global_offset][4] +
                     dist_lagging[dna_global_offset][5] +
                     dist_lagging[dna_global_offset][6] +
                     dist_lagging[dna_global_offset][7] +
                     dist_lagging[dna_global_offset][8] +
                     dist_lagging[dna_global_offset][9] +
                     dist_lagging[dna_global_offset][10] +
                     dist_lagging[dna_global_offset][11] +
                     dist_lagging[dna_global_offset][12] +
                     dist_lagging[dna_global_offset][13] +
                     dist_lagging[dna_global_offset][14] +
                     dist_lagging[dna_global_offset][15] +
                     dist_lagging[dna_global_offset][16] +
                     dist_lagging[dna_global_offset][17] +
                     dist_lagging[dna_global_offset][18] +
                     dist_lagging[dna_global_offset][19] +
                     dist_lagging[dna_global_offset][20] +
                     dist_lagging[dna_global_offset][21];

      if (dist_lag <= 4) {
        int rna_idx = atomicAdd(nb_promoters + indiv_id, 1);

        dynPromoterList[indiv_id][rna_idx].pos = dna_pos;
        dynPromoterList[indiv_id][rna_idx].leading_or_lagging = false;
        dynPromoterList[indiv_id][rna_idx].error = dist_lag;
      }
    }
  }
}


__global__
void search_start_stop_RNA_bucket(size_t* dna_size, char** dna, int8_t** dna_lead_promoter,
                             int8_t** dna_lag_promoter, int* nb_promoters,
                             pStruct** dynPromoterList,
                             int8_t** dna_lead_term, int8_t** dna_lag_term,
                             int bucket_size, int block_size) {
  int indiv_id = blockIdx.x / block_size;
  int block_id = blockIdx.x % block_size;
  int pos_block_size = blockIdx.y;

  int org_dna_pos = (gridDim.y*block_id+pos_block_size)*bucket_size;

  __shared__ int prom_dist_leading[BUCKET_MAX_SIZE][26];
  __shared__ int prom_dist_lagging[BUCKET_MAX_SIZE][26];


  __shared__ int term_dist_leading[BUCKET_MAX_SIZE][4];
  __shared__ int term_dist_lagging[BUCKET_MAX_SIZE][4];

  __shared__ int cached_dna[62];

  //__shared__ int dist_lead[BUCKET_MAX_SIZE];
  //__shared__ int dist_lag[BUCKET_MAX_SIZE];

  int motif_id = threadIdx.x % 52;
  int dna_global_offset = threadIdx.x / 52;

  int cached_dna_pos = 21+dna_global_offset;
  int dna_pos = org_dna_pos+dna_global_offset;


  if (dna_pos < dna_size[indiv_id] && dna_size[indiv_id] >= PROM_SIZE) {

    if (threadIdx.x < 62) {

      int load_pos = org_dna_pos-21+threadIdx.x;
      load_pos = load_pos < 0 ? dna_size[indiv_id] + load_pos : load_pos;
      load_pos = load_pos >= dna_size[indiv_id] ?
                     load_pos - dna_size[indiv_id] :
                     load_pos;
      //printf("Loading into shared memory at %d : DNA pos %d (w/o mod %d size %lu)\n",threadIdx.x,load_pos,org_dna_pos-21+threadIdx.x,dna_size[indiv_id]);
      cached_dna[threadIdx.x] = dna[indiv_id][load_pos];
    }



    /*if (indiv_id == 0 && blockIdx.x == 0 && blockIdx.y == 0)
      printf("Thread %d Block X %d block Y %d -- Motif ID %d (offset %d) : DNA POS %d (cached %d min %d max %d)\n",
             threadIdx.x,blockIdx.x,blockIdx.y,motif_id,dna_global_offset,dna_pos,cached_dna_pos,cached_dna_pos-21,cached_dna_pos+21);*/

    __syncthreads();

    if (motif_id >= 26 && motif_id < 48) {
      // LAGGING
      int t_motif_id = motif_id - 26;
      prom_dist_lagging[dna_global_offset][t_motif_id] =
          PROM_SEQ_LAG[t_motif_id] == cached_dna[cached_dna_pos-t_motif_id] ? 0 : 1;
    } else if (motif_id < 22) {
      // LEADING
      prom_dist_leading[dna_global_offset][motif_id] =
          PROM_SEQ_LEAD[motif_id] == cached_dna[cached_dna_pos+motif_id] ? 0 : 1;
    } else if (motif_id >= 22 && motif_id < 26) {
      int t_motif_id = motif_id - 22;
      // LEADING
      term_dist_leading[dna_global_offset][t_motif_id] =
          cached_dna[cached_dna_pos+t_motif_id] != cached_dna[cached_dna_pos-t_motif_id+10] ? 1 : 0;
    } else {
      int t_motif_id = motif_id - 48;
      term_dist_lagging[dna_global_offset][t_motif_id] =
          cached_dna[cached_dna_pos-t_motif_id] != cached_dna[cached_dna_pos+t_motif_id-10] ? 1 : 0;
    }

    __syncthreads();

    if (motif_id % 52 == 0) {

      int dist_lead = prom_dist_leading[dna_global_offset][0] +
          prom_dist_leading[dna_global_offset][1] +
          prom_dist_leading[dna_global_offset][2] +
          prom_dist_leading[dna_global_offset][3] +
          prom_dist_leading[dna_global_offset][4] +
          prom_dist_leading[dna_global_offset][5] +
          prom_dist_leading[dna_global_offset][6] +
          prom_dist_leading[dna_global_offset][7] +
          prom_dist_leading[dna_global_offset][8] +
          prom_dist_leading[dna_global_offset][9] +
          prom_dist_leading[dna_global_offset][10] +
          prom_dist_leading[dna_global_offset][11] +
          prom_dist_leading[dna_global_offset][12] +
          prom_dist_leading[dna_global_offset][13] +
          prom_dist_leading[dna_global_offset][14] +
          prom_dist_leading[dna_global_offset][15] +
          prom_dist_leading[dna_global_offset][16] +
          prom_dist_leading[dna_global_offset][17] +
          prom_dist_leading[dna_global_offset][18] +
          prom_dist_leading[dna_global_offset][19] +
          prom_dist_leading[dna_global_offset][20] +
          prom_dist_leading[dna_global_offset][21];

      if (dist_lead <= 4) {
        int rna_idx = atomicAdd(nb_promoters + indiv_id, 1);

        dynPromoterList[indiv_id][rna_idx].pos = dna_pos;
        dynPromoterList[indiv_id][rna_idx].leading_or_lagging = true;
        dynPromoterList[indiv_id][rna_idx].error = dist_lead;

//        if (indiv_id == 0) printf("New Start RNA Found ! POS %d RNA Idx %d\n",dna_pos,rna_idx);
      }
    }
    else if (motif_id % 52 == 22) {
      int dist_lead = term_dist_leading[dna_global_offset][0] +
          term_dist_leading[dna_global_offset][1] +
          term_dist_leading[dna_global_offset][2] +
          term_dist_leading[dna_global_offset][3];
      dna_lead_term[indiv_id][dna_pos] = dist_lead == 4 ? 1 : 0;
  /*    if (dna_lead_term[indiv_id][dna_pos] == 4)
        printf("New STOP RNA Found ! POS %d RNA Idx %d\n",dna_pos);*/
    }
    else if (motif_id % 52 == 26) {
      int dist_lag = prom_dist_lagging[dna_global_offset][0] +
          prom_dist_lagging[dna_global_offset][1] +
          prom_dist_lagging[dna_global_offset][2] +
          prom_dist_lagging[dna_global_offset][3] +
          prom_dist_lagging[dna_global_offset][4] +
          prom_dist_lagging[dna_global_offset][5] +
          prom_dist_lagging[dna_global_offset][6] +
          prom_dist_lagging[dna_global_offset][7] +
          prom_dist_lagging[dna_global_offset][8] +
          prom_dist_lagging[dna_global_offset][9] +
          prom_dist_lagging[dna_global_offset][10] +
          prom_dist_lagging[dna_global_offset][11] +
          prom_dist_lagging[dna_global_offset][12] +
          prom_dist_lagging[dna_global_offset][13] +
          prom_dist_lagging[dna_global_offset][14] +
          prom_dist_lagging[dna_global_offset][15] +
          prom_dist_lagging[dna_global_offset][16] +
          prom_dist_lagging[dna_global_offset][17] +
          prom_dist_lagging[dna_global_offset][18] +
          prom_dist_lagging[dna_global_offset][19] +
          prom_dist_lagging[dna_global_offset][20] +
          prom_dist_lagging[dna_global_offset][21];

      if (dist_lag <= 4) {
        int rna_idx = atomicAdd(nb_promoters + indiv_id, 1);

        dynPromoterList[indiv_id][rna_idx].pos = dna_pos;
        dynPromoterList[indiv_id][rna_idx].leading_or_lagging = false;
        dynPromoterList[indiv_id][rna_idx].error = dist_lag;
      }
    }
    else if (motif_id % 52 == 48) {
      int dist_lag = term_dist_lagging[dna_global_offset][0] +
          term_dist_lagging[dna_global_offset][1] +
          term_dist_lagging[dna_global_offset][2] +
          term_dist_lagging[dna_global_offset][3];
      dna_lag_term[indiv_id][dna_pos] = dist_lag == 4 ? 1 : 0;
    }
  }
}


__global__
void search_stop_RNA(size_t* dna_size, char** dna, int8_t** dna_lead_term, int8_t** dna_lag_term, int block_size) {

  int indiv_id = blockIdx.x / block_size;
  int block_id = blockIdx.x % block_size;
  int pos_block_size = blockIdx.y;

  int dna_pos = gridDim.y * block_id + pos_block_size;

  __shared__ int dist_leading[4];
  __shared__ int dist_lagging[4];

  int motif_id = threadIdx.x;

  if (dna_size[indiv_id] < PROM_SIZE) {
    //printf("STOP -- SMALL SIZE\n");
    /*if (dna_pos < dna_size[indiv_id] && threadIdx.x == 0) {
      dna_lead_term[indiv_id][dna_pos] = 0;
      dna_lag_term[indiv_id][dna_pos] = 0;
    }*/
  } else if (dna_pos < dna_size[indiv_id]) {

    if (motif_id >= 4) {
      // LAGGING
      motif_id -= 4;
      int pos_1 = dna_pos - motif_id < 0 ?
                  dna_size[indiv_id] + (dna_pos - motif_id) : dna_pos -
                                                              motif_id;
      int pos_2 = dna_pos + motif_id - 10 < 0 ?
                  dna_size[indiv_id] + (dna_pos + motif_id - 10) : dna_pos +
                                                                   motif_id -
                                                                   10;

      dist_lagging[motif_id] =
          dna[indiv_id][pos_1] != dna[indiv_id][pos_2] ? 1 : 0;
    } else {
      // LEADING
      int pos_1 = dna_pos + motif_id >= dna_size[indiv_id] ?
                  (dna_pos + motif_id) - dna_size[indiv_id] : dna_pos +
                                                              motif_id;
      int pos_2 = dna_pos - motif_id + 10 >= dna_size[indiv_id] ?
                  10 + dna_pos - motif_id - dna_size[indiv_id] : dna_pos -
                                                                 motif_id +
                                                                 10;
      dist_leading[motif_id] =
          dna[indiv_id][pos_1] != dna[indiv_id][pos_2] ? 1 : 0;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
      int dist_lead = dist_leading[0] + dist_leading[1] + dist_leading[2] +
                      dist_leading[3];
      int dist_lag = dist_lagging[0] + dist_lagging[1] + dist_lagging[2] +
                     dist_lagging[3];

      dna_lead_term[indiv_id][dna_pos] = dist_lead == 4 ? 1 : 0;
      dna_lag_term[indiv_id][dna_pos] = dist_lag == 4 ? 1 : 0;

      //printf("%d Found STOP at %d\n",indiv_id,dna_pos);

    }
  }
}

__global__
void internal_init_RNA_struct(cRNA*** rna, int32_t* max_nb_elements_rna_list, int indiv_id) {
  int offset = blockIdx.x*1024;
  int rna_idx = offset+threadIdx.x;

  if (rna_idx < max_nb_elements_rna_list[indiv_id]) {
    rna[indiv_id][rna_idx] = (cRNA*) malloc(sizeof(cRNA));
    rna[indiv_id][rna_idx]->max_protein_elements = 200;

    rna[indiv_id][rna_idx]->start_prot = (uint32_t*) malloc(
        rna[indiv_id][rna_idx]->max_protein_elements * sizeof(uint32_t));
  }
}

__global__
void init_RNA_struct(int pop_size, cRNA*** rna, int* nb_promoters, int32_t* max_nb_rna,int32_t* idx_rna, int32_t* max_nb_elements_rna_list) {
  int indiv_id = blockIdx.x;

  if (nb_promoters[indiv_id] > 0) {
    if (nb_promoters[indiv_id] >= max_nb_elements_rna_list[indiv_id]) {
      // Increase RNA List size
      for (int i=0; i < max_nb_elements_rna_list[indiv_id];i++) {
        free(rna[indiv_id][i]->start_prot);
        free(rna[indiv_id][i]);
      }

      free(rna[indiv_id]);
      int before_cpt=max_nb_elements_rna_list[indiv_id];
      max_nb_elements_rna_list[indiv_id] = (1+((int32_t)nb_promoters[indiv_id]/RNA_LIST_INCR_SIZE))*RNA_LIST_INCR_SIZE;

      rna[indiv_id] = (cRNA**) malloc((max_nb_elements_rna_list[indiv_id]+1)*sizeof(cRNA*));


      int block_offset = max_nb_elements_rna_list[indiv_id]/1024 + 1;

      internal_init_RNA_struct<<<block_offset,1024>>>(rna,max_nb_elements_rna_list,indiv_id);
      //hipDeviceSynchronize();
      //
      /*for (int i=0; i < max_nb_elements_rna_list[indiv_id]; i++) {
        rna[indiv_id][i] = (cRNA*) malloc(sizeof(cRNA));
        //printf("Malloc POINTER RNA %d indiv %d : %p\n",max_nb_elements_rna_list[indiv_id],indiv_id,l_rna);
        rna[indiv_id][i]->max_protein_elements = 250;//RNA_LIST_PROTEIN_INCR_SIZE;

        rna[indiv_id][i]->start_prot =  (int*) malloc(
            rna[indiv_id][i]->max_protein_elements  * sizeof(int));
      }*/
/*      printf("Malloc Increase DONE RNA %d indiv %d (before %d current %d)\n",max_nb_elements_rna_list[indiv_id],indiv_id,
             before_cpt,nb_promoters[indiv_id]);*/
    } else if (nb_promoters[indiv_id] < max_nb_elements_rna_list[indiv_id]/2 && max_nb_elements_rna_list[indiv_id] - RNA_LIST_INCR_SIZE > 0) {
      // Decrease RNA List size
      for (int i=0; i < max_nb_elements_rna_list[indiv_id];i++) {
        free(rna[indiv_id][i]->start_prot);
        free(rna[indiv_id][i]);
      }
      free(rna[indiv_id]);
      //max_nb_elements_rna_list[indiv_id] -= RNA_LIST_INCR_SIZE;
      int before_cpt=max_nb_elements_rna_list[indiv_id];
      max_nb_elements_rna_list[indiv_id]  = max_nb_elements_rna_list[indiv_id] - RNA_LIST_INCR_SIZE == 0 ?
                                            RNA_LIST_INCR_SIZE :  max_nb_elements_rna_list[indiv_id] - RNA_LIST_INCR_SIZE;

      rna[indiv_id] = (cRNA**) malloc((max_nb_elements_rna_list[indiv_id]+1)*sizeof(cRNA*));

      int block_offset = max_nb_elements_rna_list[indiv_id]/1024 + 1;

      internal_init_RNA_struct<<<block_offset,1024>>>(rna,max_nb_elements_rna_list,indiv_id);
      //hipDeviceSynchronize();

      /*for (int i=0; i < max_nb_elements_rna_list[indiv_id];i++) {
        rna[indiv_id][i] = (cRNA*) malloc(sizeof(cRNA));
        rna[indiv_id][i]->max_protein_elements = 250;
        rna[indiv_id][i]->start_prot =  (int*) malloc(
            rna[indiv_id][i]->max_protein_elements  * sizeof(int));
        //RNA_LIST_PROTEIN_INCR_SIZE;
      }*/
      /*printf("Malloc Decrease DONE RNA %d indiv %d (before %d current %d)\n",max_nb_elements_rna_list[indiv_id],indiv_id,
              before_cpt,nb_promoters[indiv_id]);*/
    }

    atomicMax(max_nb_rna,nb_promoters[indiv_id]+1);
    idx_rna[indiv_id] = 0;
  }
}

__global__
void compute_RNA(int pop_size, int8_t** dna_lead_promoter,
                 int8_t** dna_lag_promoter, int8_t** dna_lead_term,
                 int8_t** dna_lag_term, char** dna, size_t* dna_size,
                 cRNA*** rna, int32_t* idx_rna,  int* nb_promoters,
                 pStruct** dynPromoterList, int threads_size, int thread_dim) {

  int indiv_id = blockIdx.x / threads_size;
  int block_id = blockIdx.x % threads_size;
  int pos_block_size = threadIdx.x;

  int rna_idx = thread_dim*block_id+pos_block_size;

  if (dna_size[indiv_id] >= PROM_SIZE && rna_idx < nb_promoters[indiv_id]) {
    if (dynPromoterList[indiv_id][rna_idx].leading_or_lagging) {
      // LEADING
      // Search for terminator
      int k = dynPromoterList[indiv_id][rna_idx].pos + 22;
      k = k >= dna_size[indiv_id] ? k - dna_size[indiv_id] : k;
      int k_end = k;
      do {
        if (dna_lead_term[indiv_id][k] == 1) {
          int32_t rna_end =
              k + 10 >= dna_size[indiv_id] ? k + 10 - dna_size[indiv_id] :
              k +
              10;

          int32_t rna_length = 0;

          if (dynPromoterList[indiv_id][rna_idx].pos > rna_end)
            rna_length = dna_size[indiv_id] - dynPromoterList[indiv_id][rna_idx].pos + rna_end;
          else
            rna_length = rna_end - dynPromoterList[indiv_id][rna_idx].pos;

          if (rna_length < 19) {
            break;
          }

          int idx = atomicAdd(idx_rna + indiv_id, 1);

          rna[indiv_id][idx]->begin = dynPromoterList[indiv_id][rna_idx].pos;
          rna[indiv_id][idx]->end = rna_end;
          rna[indiv_id][idx]->length = rna_length;
          rna[indiv_id][idx]->leading_lagging = !dynPromoterList[indiv_id][rna_idx].leading_or_lagging;

          rna[indiv_id][idx]->e = 1.0 -
                     fabs(((float) dynPromoterList[indiv_id][rna_idx].error)) /
                     5.0;

          /*if (rna_length > rna[indiv_id][idx]->max_protein_elements) {
            // Increase size
            free(rna[indiv_id][idx]->start_prot);
            int before_cpt = rna[indiv_id][idx]->max_protein_elements;
            rna[indiv_id][idx]->max_protein_elements=(1+((int32_t)rna_length/RNA_LIST_PROTEIN_INCR_SIZE))*RNA_LIST_PROTEIN_INCR_SIZE;
            rna[indiv_id][idx]->start_prot = (uint32_t*) malloc(
                (rna[indiv_id][idx]->max_protein_elements + 1) * sizeof(uint32_t));
            printf("Malloc Increase DONE RNA Protein List %d indiv %d -- %d (before %d current %d)\n",rna_length,
                   indiv_id,rna_idx,
                   before_cpt,rna[indiv_id][idx]->max_protein_elements);
          } else if ((rna_length < rna[indiv_id][idx]->max_protein_elements/2) && (rna[indiv_id][idx]->max_protein_elements - RNA_LIST_PROTEIN_INCR_SIZE > 0)) {
            // Decrease size
            free(rna[indiv_id][idx]->start_prot);
            int before_cpt = rna[indiv_id][idx]->max_protein_elements;
            rna[indiv_id][idx]->max_protein_elements-=RNA_LIST_PROTEIN_INCR_SIZE;
            rna[indiv_id][idx]->start_prot = (int*) malloc(
                (rna[indiv_id][idx]->max_protein_elements + 1) * sizeof(int));
            if (indiv_id == 828)  printf("Malloc Decrease DONE RNA Protein List %d indiv %d -- %d (before %d current %d)\n",rna_length,
                   indiv_id,rna_idx,
                   before_cpt,rna[indiv_id][idx]->max_protein_elements);
          }*/

          rna[indiv_id][idx]->start_lenght = 1;
          rna[indiv_id][idx]->nb_protein = 0;

          break;
        }

        k++;
        k = k >= dna_size[indiv_id] ? k - dna_size[indiv_id] : k;
      } while (k != k_end);

    } else {
      // LAGGING

      // Search for terminator
      int k = dynPromoterList[indiv_id][rna_idx].pos - 22;
      k = k < 0 ? dna_size[indiv_id] + k : k;
      int k_end = k;
      do {

        if (dna_lag_term[indiv_id][k] == 1) {
          int32_t rna_end = k - 10 < 0 ? dna_size[indiv_id] + (k - 10) : k - 10;

          int32_t rna_length = 0;

          if (dynPromoterList[indiv_id][rna_idx].pos < rna_end)
            rna_length = dynPromoterList[indiv_id][rna_idx].pos + dna_size[indiv_id] - rna_end;
          else
            rna_length = dynPromoterList[indiv_id][rna_idx].pos - rna_end;

          if (rna_length < 19) {
            break;
          }

          int idx = atomicAdd(idx_rna + indiv_id, 1);

          /*printf("Indiv %d -- Setting RNA %d to begin at %d (promoter idx %d out of %d)\n",
                 indiv_id,idx,dynPromoterList[indiv_id][rna_idx].pos,rna_idx,nb_promoters[indiv_id]);*/

          rna[indiv_id][idx]->begin = dynPromoterList[indiv_id][rna_idx].pos;
          rna[indiv_id][idx]->end = rna_end;
          rna[indiv_id][idx]->length = rna_length;

          rna[indiv_id][idx]->leading_lagging = !dynPromoterList[indiv_id][rna_idx].leading_or_lagging;


          rna[indiv_id][idx]->e =
              1.0 - ((float) dynPromoterList[indiv_id][rna_idx].error) / 5.0;


          /*if (rna_length > rna[indiv_id][idx]->max_protein_elements) {
            // Increase size
            free(rna[indiv_id][idx]->start_prot);
            int before_cpt = rna[indiv_id][idx]->max_protein_elements;
            rna[indiv_id][idx]->max_protein_elements=(1+((int32_t)rna_length/RNA_LIST_PROTEIN_INCR_SIZE))*RNA_LIST_PROTEIN_INCR_SIZE;
            rna[indiv_id][idx]->start_prot = (uint32_t*) malloc(
                (rna[indiv_id][idx]->max_protein_elements + 1) * sizeof(uint32_t));
            printf("Malloc Increase DONE RNA Protein List %d indiv %d -- %d (before %d current %d)\n",
                   rna_length,
                   indiv_id,rna_idx,
                   before_cpt,rna[indiv_id][idx]->max_protein_elements);
          } else if ((rna_length < rna[indiv_id][idx]->max_protein_elements/2) && (rna[indiv_id][idx]->max_protein_elements - RNA_LIST_PROTEIN_INCR_SIZE > 0)) {
            // Decrease size
            free(rna[indiv_id][idx]->start_prot);
            int before_cpt = rna[indiv_id][idx]->max_protein_elements;
            rna[indiv_id][idx]->max_protein_elements-=RNA_LIST_PROTEIN_INCR_SIZE;
            rna[indiv_id][idx]->start_prot = (int*) malloc(
                (rna[indiv_id][idx]->max_protein_elements + 1) * sizeof(int));
            if (indiv_id == 828)   printf("Malloc Decrease DONE RNA Protein List %d indiv %d -- %d (before %d current %d)\n",
                   rna_length,
                   indiv_id,rna_idx,
                   before_cpt,rna[indiv_id][idx]->max_protein_elements);
          }*/

          rna[indiv_id][idx]->start_lenght = 1;
          rna[indiv_id][idx]->nb_protein = 0;

          break;
        }

        k--;
        k = k < 0 ? dna_size[indiv_id] + k : k;
      } while (k != k_end);

    }
  }
}

__global__ void max_rna(int32_t* idx_rna, int32_t* max_nb_rna) {
  int indiv_id = blockIdx.x;

  atomicMax(max_nb_rna, idx_rna[indiv_id]);
}

__global__ void compute_start_protein(int32_t* idx_rna, cRNA*** rna,
                                      char** dna,size_t* dna_size, int32_t* nb_protein, int threads_size, int thread_dim) {
//  int indiv_id = blockIdx.x;
//  int rna_idx = threadIdx.x;

  int indiv_id = blockIdx.x / threads_size;
  int block_id = blockIdx.x % threads_size;
  int pos_block_size = threadIdx.x;

  int rna_idx = thread_dim*block_id+pos_block_size;


  /*if (indiv_id == 737 && rna_idx == 47)
    printf("Searching for protein start on %d (%d %d) with %d\n",rna_idx,indiv_id,block_id,pos_block_size);*/

  if (rna_idx < idx_rna[indiv_id]) {

    int c_pos = rna[indiv_id][rna_idx]->begin;
    if (rna[indiv_id][rna_idx]->length > 22) {
      if (rna[indiv_id][rna_idx]->leading_lagging == 0) {
        c_pos += 22;
        c_pos =
            c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
      } else {
        c_pos -= 22;
        //if (indiv_id == 915 && rna_idx < 2) printf("C_POS -22 : %d\n", c_pos);
        c_pos = c_pos < 0 ? ((int) dna_size[indiv_id]) + c_pos : c_pos;
        /*if (indiv_id == 737 && rna_idx == 47)
          printf("MOD C_POS -22 : %d SIZE - MOD : %d SIZE: %lu\n",
                 c_pos, (int) (dna_size[indiv_id] - c_pos),
                 dna_size[indiv_id]);*/

      }

      // TODO IF SIZE is smaller than 9 return

      /*if (indiv_id == 737) {
        printf("Starting to search in between %d (%d -- %lu) and %d\n",
               c_pos, rna[indiv_id][rna_idx]->begin, dna_size[indiv_id],
               rna[indiv_id][rna_idx]->end);
      }*/
      while (c_pos != rna[indiv_id][rna_idx]->end) {

        bool start = false;
        int t_pos, k_t;

        if (rna[indiv_id][rna_idx]->leading_lagging == 0) {
          // Search for Shine Dalgarro + START codon on LEADING
          for (int k = 0; k < 9; k++) {
            k_t = k >= 6 ? k + 4 : k;
            t_pos = c_pos + k_t >= dna_size[indiv_id] ? c_pos + k_t -
                                                        dna_size[indiv_id] :
                    c_pos + k_t;

            if (dna[indiv_id][t_pos] == SHINE_DAL_SEQ_LEAD[k]) {
              start = true;
            } else {
              start = false;
              break;
            }
          }

        } else {
          // Search for Shine Dalgarro + START codon on LAGGING
          for (int k = 0; k < 9; k++) {
            k_t = k >= 6 ? k + 4 : k;
            t_pos =
                c_pos - k_t < 0 ? dna_size[indiv_id] - c_pos - k_t : c_pos -
                                                                     k_t;

            /*if (indiv_id == 737 && rna_idx == 47)
              printf("Search protein start at %d : %d %d -- %c / %c\n",c_pos,t_pos,k_t,
                     dna[indiv_id][t_pos],SHINE_DAL_SEQ_LAG[k]);*/

            if (dna[indiv_id][t_pos] == SHINE_DAL_SEQ_LAG[k]) {
              start = true;
            } else {
              start = false;
              break;
            }
          }
        }

        if (start) {
          int prot_idx = atomicAdd(&(rna[indiv_id][rna_idx]->nb_protein), 1);

          rna[indiv_id][rna_idx]->start_prot[prot_idx] = c_pos;

          /*if (indiv_id == 737 && rna_idx == 47)
            printf("RNA : %d %d || Current : %d -- %lu\n",
                   rna[indiv_id][rna_idx]->begin,
                   rna[indiv_id][rna_idx]->end,
                   c_pos, dna_size[indiv_id]);*/


          atomicAdd(&nb_protein[indiv_id], 1);
        }

        if (rna[indiv_id][rna_idx]->leading_lagging == 0) {
          c_pos++;
          c_pos =
              c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
        } else {
          c_pos--;
          c_pos = c_pos < 0 ? dna_size[indiv_id] + c_pos : c_pos;
        }

        /*if (indiv_id == 915 && rna_idx < 2) {
          printf("POS %d END %d\n", c_pos, rna[indiv_id][rna_idx]->end);
        }*/
      }
    } /*else if (indiv_id == 915 && rna_idx < 2) {
      printf("TOO SMALL !!!\n");
    }*/
  }


}

__global__ void init_protein_struct(int pop_size, int32_t* nb_protein,
                                    cProtein** protein_list, int32_t* idx_protein,
                                    cRNA*** rna, int32_t* idx_rna, int32_t* max_nb_protein, int* max_nb_elements_protein_list,
                                    int threads_size, int thread_dim) {
  //int indiv_id = blockIdx.x;
  //int rna_idx = threadIdx.x;

  int indiv_id = blockIdx.x / threads_size;
  int block_id = blockIdx.x % threads_size;
  int pos_block_size = threadIdx.x;

  int rna_idx = thread_dim*block_id+pos_block_size;

  if (rna_idx == 0 && nb_protein[indiv_id] > 0) {
    //printf("%d -- Number of Protein %d (array size %d)\n",indiv_id,nb_protein[indiv_id],max_nb_elements_protein_list[indiv_id]);
    /*if (max_nb_elements_protein_list[indiv_id] == 0) {
      max_nb_elements_protein_list[indiv_id] = (1+((int32_t)nb_protein[indiv_id]/PROTEIN_LIST_INCR_SIZE))*PROTEIN_LIST_INCR_SIZE;

      protein_list[indiv_id] = (cProtein*) malloc((max_nb_elements_protein_list[indiv_id]+1)*sizeof(cProtein));

      printf("Malloc Init DONE Protein %d indiv %d (current %d)\n",max_nb_elements_protein_list[indiv_id],indiv_id,
             nb_protein[indiv_id]);
    } else */if (nb_protein[indiv_id] >= max_nb_elements_protein_list[indiv_id]) {
      free(protein_list[indiv_id]);
      int before_cpt=max_nb_elements_protein_list[indiv_id];
      max_nb_elements_protein_list[indiv_id] = (1+((int32_t)nb_protein[indiv_id]/PROTEIN_LIST_INCR_SIZE))*PROTEIN_LIST_INCR_SIZE;

      protein_list[indiv_id] = (cProtein*) malloc((max_nb_elements_protein_list[indiv_id]+1)*sizeof(cProtein));

      /*printf("Malloc Increase DONE Protein %d indiv %d (before %d current %d)\n",max_nb_elements_protein_list[indiv_id],indiv_id,
             before_cpt,nb_protein[indiv_id]);*/
    } else if (nb_protein[indiv_id] < max_nb_elements_protein_list[indiv_id]/2 && (max_nb_elements_protein_list[indiv_id] - PROTEIN_LIST_INCR_SIZE > 0)) {
      free(protein_list[indiv_id]);
      //max_nb_elements_rna_list[indiv_id] -= RNA_LIST_INCR_SIZE;
      int before_cpt=max_nb_elements_protein_list[indiv_id];
      max_nb_elements_protein_list[indiv_id]  = max_nb_elements_protein_list[indiv_id] - PROTEIN_LIST_INCR_SIZE == 0 ?
                                            PROTEIN_LIST_INCR_SIZE :  max_nb_elements_protein_list[indiv_id] - PROTEIN_LIST_INCR_SIZE;

      protein_list[indiv_id] = (cProtein*) malloc((max_nb_elements_protein_list[indiv_id]+1)*sizeof(cProtein));

      /*printf("Malloc Decrease DONE Protein %d indiv %d (before %d current %d)\n",max_nb_elements_protein_list[indiv_id],indiv_id,
             before_cpt,nb_protein[indiv_id]);*/
    }
    //printf("%d -- END OF Number of Protein %d (array size %d)\n",indiv_id,nb_protein[indiv_id],max_nb_elements_protein_list[indiv_id]);

    //protein_list[indiv_id] = (cProtein**) malloc((nb_protein[indiv_id])*sizeof(cProtein*));
    idx_protein[indiv_id] = 0;
  }

  if (rna_idx < idx_rna[indiv_id])
    atomicMax(max_nb_protein,nb_protein[indiv_id]);
}

__global__ void display_size_dna(size_t* dna_size) {
  int indiv_id = blockIdx.x;

  //cProtein* l_protein = (cProtein*) malloc(sizeof(22));
  //if (indiv_id ==0) printf("Address : %lu\n", (unsigned long) l_protein);
}

__global__ void compute_protein(cRNA*** rna, cProtein** protein_list, int32_t* idx_protein,
                                size_t* dna_size,char** dna,int32_t* idx_rna, int threads_size, int thread_dim, int block_size) {
  //int indiv_id = blockIdx.x;

  int debug_iid = 915;
  //int rna_idx = blockIdx.y;

  //int protein_idx = threadIdx.x;

  int indiv_id = blockIdx.x / block_size;
  int block_id = blockIdx.x % block_size;
  int pos_block_size = blockIdx.y;

  int block_x = gridDim.y*block_id+pos_block_size;

  int rna_idx = block_x / threads_size;
  block_id = block_x % threads_size;
  pos_block_size = threadIdx.x;

  int protein_idx = thread_dim*block_id+pos_block_size;

  if (rna_idx < idx_rna[indiv_id])
    if (protein_idx < rna[indiv_id][rna_idx]->nb_protein) {
      int start_protein_pos = rna[indiv_id][rna_idx]->leading_lagging == 0 ? rna[indiv_id][rna_idx]->start_prot[protein_idx] + 13 : rna[indiv_id][rna_idx]->start_prot[protein_idx] - 13;
      int length = -1;

      if (rna[indiv_id][rna_idx]->leading_lagging == 0) {
        start_protein_pos = start_protein_pos >= dna_size[indiv_id] ?
                            start_protein_pos - dna_size[indiv_id]
                                                                    : start_protein_pos;

        if (rna[indiv_id][rna_idx]->start_prot[protein_idx] < rna[indiv_id][rna_idx]->end) {
          length = rna[indiv_id][rna_idx]->end - rna[indiv_id][rna_idx]->start_prot[protein_idx];
        } else {
          length = dna_size[indiv_id] - rna[indiv_id][rna_idx]->start_prot[protein_idx] + rna[indiv_id][rna_idx]->end + 1;
        }

        length -= 13;

        /*if (indiv_id == 737 && rna_idx == 47)
          printf("----> START %d %d END %d LENGTH %d Indiv ID %d RNA IDX %d (block.X %d block_x %d "
                     "block.Y %d pos_block %d) Protein IDX %d (block id %d pos block size %d threads size %d)\n",rna[indiv_id][rna_idx]->start_prot[protein_idx],start_protein_pos,
                 rna[indiv_id][rna_idx]->end,length,indiv_id,rna_idx,blockIdx.x,block_x,blockIdx.y,
                 pos_block_size,protein_idx,block_id,pos_block_size,threads_size);*/

      } else {
        start_protein_pos = start_protein_pos < 0 ?
                            dna_size[indiv_id] + start_protein_pos
                                                  : start_protein_pos;

        if (rna[indiv_id][rna_idx]->start_prot[protein_idx] > rna[indiv_id][rna_idx]->end) {
          length = rna[indiv_id][rna_idx]->start_prot[protein_idx] - rna[indiv_id][rna_idx]->end;
        } else {
          length = rna[indiv_id][rna_idx]->start_prot[protein_idx] +  dna_size[indiv_id] - rna[indiv_id][rna_idx]->end;
        }


        length -= 13;
        /*if (indiv_id == 737 && rna_idx == 47)
          printf("----> START %d %d END %d LENGTH %d Indiv ID %d RNA IDX %d (block.X %d block_x %d "
                     "block.Y %d pos_block %d) Protein IDX %d (block id %d pos block size %d threads size %d)\n",rna[indiv_id][rna_idx]->start_prot[protein_idx],start_protein_pos,
                 rna[indiv_id][rna_idx]->end,length,indiv_id,rna_idx,blockIdx.x,block_x,blockIdx.y,
                 pos_block_size,protein_idx,block_id,pos_block_size,threads_size);*/
      }

      bool is_protein = false;

      /*if (indiv_id == debug_iid) {
        printf("LENGTH is %d\n",length);
      }*/
      length+=1;
      length = length - (length%3);
      /*if (indiv_id == debug_iid) {
        printf("LENGTH UPDATED is %d\n",length);
      }*/

      for (int loop_i = 0; length - loop_i >= 2; loop_i+=3) {//start_protein_pos != rna[indiv_id][rna_idx]->end) {
        int t_k;

        /*if (indiv_id == debug_iid)
          printf("Starting loop id %d\n",loop_i);*/

        if (rna[indiv_id][rna_idx]->leading_lagging == 0) {
          start_protein_pos = start_protein_pos >= dna_size[indiv_id] ?
                              start_protein_pos - dna_size[indiv_id]
                                                                      : start_protein_pos;
          is_protein=false;


          /*if (indiv_id == debug_iid)
            printf("Starting search at %d\n",start_protein_pos);*/

          for (int k = 0; k < 3; k++) {
            t_k = start_protein_pos+k >= dna_size[indiv_id] ?
                  start_protein_pos - dna_size[indiv_id] + k :
                  start_protein_pos + k;

            /*printf("%d-%d-%d :: %lu : %d || %c (%d)\n", // %c (%d) /
                   indiv_id,rna_idx,protein_idx,
                   dna_size[indiv_id],
                   t_k,
            //       dna[indiv_id][t_k],t_k,
                   PROTEIN_END_LEAD[k],k);*/

            if (dna[indiv_id][t_k] == PROTEIN_END_LEAD[k]) {
              //ab=1;
              is_protein=true;
            } else {
              //ab=0;
              is_protein=false;
              break;
            }
          }

          /*if (indiv_id == 737 && rna_idx == 47)
            printf("Protein %d : %d : %d : %d : %d -> %d |||| %d %d --  E %d ERNA %d\n",indiv_id,rna_idx,protein_idx,loop_i,
                   start_protein_pos,is_protein,
                   length - loop_i,length,t_k,rna[indiv_id][rna_idx]->end);*/
          //printf("Protein %d : %d : %d : %d : %d -> %d (%d)\n",indiv_id,rna_idx,protein_idx,loop_i,start_protein_pos,is_protein,ab);

          //cProtein* test = (cProtein*) malloc(20);
          //cRNA* l_rna = (cRNA*)malloc(sizeof(cRNA));
          if (is_protein ) {
            int prot_length = -1;
            if (rna[indiv_id][rna_idx]->start_prot[protein_idx]+13 < t_k) {
              prot_length = t_k - (rna[indiv_id][rna_idx]->start_prot[protein_idx]+13);
            } else {
              prot_length = dna_size[indiv_id] - (rna[indiv_id][rna_idx]->start_prot[protein_idx]+13) + t_k;
            }

            if (prot_length >= 3) {
              //cProtein* l_protein = (cProtein*) malloc(sizeof(cProtein));
              //cProtein* l_protein = (cProtein*)malloc(sizeof(cProtein));
              //cRNA* l_rna = (cRNA*)malloc(sizeof(cRNA));
              int idx = atomicAdd(idx_protein + indiv_id, 1);

              protein_list[indiv_id][idx].protein_start = rna[indiv_id][rna_idx]->start_prot[protein_idx];
              protein_list[indiv_id][idx].protein_end = t_k;
              protein_list[indiv_id][idx].e = rna[indiv_id][rna_idx]->e;
              protein_list[indiv_id][idx].leading_lagging = rna[indiv_id][rna_idx]->leading_lagging;
              protein_list[indiv_id][idx].protein_length = prot_length;

              /*if (indiv_id == 737)
                printf("Address %d-%d-%d -- %d : %p (%d %d) -- (%d %d) -- %d\n",
                       indiv_id, rna_idx, protein_idx, idx, (void*) l_protein,
                       l_protein->protein_start, l_protein->protein_end,
                       rna[indiv_id][rna_idx]->begin,
                       rna[indiv_id][rna_idx]->end, prot_length);*/
              // = l_protein;
            }
            break;
          }



          start_protein_pos+=3;
          start_protein_pos = start_protein_pos >= dna_size[indiv_id] ?
                                start_protein_pos - dna_size[indiv_id]
                                                                       : start_protein_pos;

          //if (indiv_id == debug_iid) printf("New position %d %d LOOPSIZE %d\n",start_protein_pos,loop_i,length-loop_i);
        } else {

          is_protein=false;
          start_protein_pos = start_protein_pos < 0 ?
                              dna_size[indiv_id] + start_protein_pos
                                                    : start_protein_pos;



          for (int k = 0; k < 3; k++) {
            t_k = start_protein_pos-k < 0 ?
                  dna_size[indiv_id] + (start_protein_pos - k) :
                  start_protein_pos - k;

            /*if (indiv_id == 1020) printf("%d-%d-%d :: %lu : %d || %c (%d) // %c (%d)\n", //
                   indiv_id,rna_idx,protein_idx,
                   dna_size[indiv_id],
                   t_k,
                   dna[indiv_id][t_k],t_k,
                  PROTEIN_END_LAG[k],k);*/

            if (dna[indiv_id][t_k] == PROTEIN_END_LAG[k]) {
              //ab=1;
                is_protein=true;
            } else {
              //ab=0;
                is_protein=false;
                break;
            }
          }


          /*if (indiv_id == debug_iid)
           printf("Protein %d : %d : %d : %d : %d -> %d |||| %d %d\n",indiv_id,rna_idx,protein_idx,loop_i,
                  start_protein_pos,is_protein,
                  length - loop_i,length);*/

          //cProtein* test = (cProtein*) malloc(20);
          //cRNA* l_rna = (cRNA*)malloc(sizeof(cRNA));

          if (is_protein) {
            int prot_length=-1;
            if (rna[indiv_id][rna_idx]->start_prot[protein_idx]-13 > t_k) {
              prot_length = (rna[indiv_id][rna_idx]->start_prot[protein_idx]-13) - t_k;
            } else {
              prot_length = (rna[indiv_id][rna_idx]->start_prot[protein_idx]-13) +  dna_size[indiv_id] - t_k;
            }

            if (prot_length >= 3) {
              //cProtein* l_protein = (cProtein*) malloc(sizeof(cProtein));
              int idx = atomicAdd(idx_protein + indiv_id, 1);
              //cProtein* l_protein = (cProtein*) malloc(sizeof(int));
              protein_list[indiv_id][idx].protein_start = rna[indiv_id][rna_idx]->start_prot[protein_idx];
              protein_list[indiv_id][idx].protein_end = t_k;

              /*if (indiv_id == 878) {
                for (int k = 0; k < 3; k++) {
                  t_k = start_protein_pos - k < 0 ?
                        dna_size[indiv_id] - start_protein_pos - k :
                        start_protein_pos - k;
                  printf("[%d -- %c] ",t_k,dna[indiv_id][t_k]);
                }
                printf("\n");
              }*/
              protein_list[indiv_id][idx].protein_length = prot_length;
              protein_list[indiv_id][idx].e = rna[indiv_id][rna_idx]->e;
              protein_list[indiv_id][idx].leading_lagging = rna[indiv_id][rna_idx]->leading_lagging;



              /*if (indiv_id == 737)
                printf("Address %d-%d-%d -- %d : %p (%d %d) -- (%d %d) -- %d %lu\n",
                       indiv_id, rna_idx, protein_idx, idx, (void*) l_protein,
                       l_protein->protein_start, l_protein->protein_end,
                       rna[indiv_id][rna_idx]->begin,
                       rna[indiv_id][rna_idx]->end, prot_length,dna_size[indiv_id]);*/
              //protein_list[indiv_id][idx] = l_protein;
            }
            break;
          }
          //if (is_protein) cProtein* l_protein = (cProtein*) malloc(1*sizeof(cProtein));
          //int* test = (int*)malloc(sizeof(int));


          start_protein_pos = start_protein_pos-3;
          start_protein_pos = start_protein_pos < 0 ?
                                dna_size[indiv_id] + start_protein_pos
                                                      : start_protein_pos;

          //if (indiv_id == debug_iid) printf("New position %d %d\n",start_protein_pos,loop_i);
        }
      }
    }
}

__global__ void max_protein(int32_t* max_nb_protein, int32_t* idx_protein) {
  int indiv_id = blockIdx.x;

  atomicMax(max_nb_protein, idx_protein[indiv_id]-1);
}

__global__ void translate_protein(float w_max, int32_t* idx_protein,
                                  cProtein** protein_list,
                                  char** dna, size_t* dna_size, int threads_size, int thread_dim) {
  int indiv_id = blockIdx.x / threads_size;
  int block_id = blockIdx.x % threads_size;
  int pos_block_size = threadIdx.x;

  int protein_idx = thread_dim*block_id+pos_block_size;

  if (protein_idx < idx_protein[indiv_id]) {
    // Translate RNA to codon
    int c_pos = protein_list[indiv_id][protein_idx].protein_start, t_pos;
    int end_pos = protein_list[indiv_id][protein_idx].protein_end;
    if (protein_list[indiv_id][protein_idx].leading_lagging == 0) {
      c_pos += 13;
      end_pos -=3;

      c_pos = c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
      end_pos = end_pos < 0 ? dna_size[indiv_id] + end_pos : end_pos;
    } else {
      c_pos -= 13;
      end_pos +=3;

      end_pos = end_pos >= dna_size[indiv_id] ? end_pos - dna_size[indiv_id] : end_pos;
      c_pos = c_pos < 0 ? dna_size[indiv_id] + c_pos : c_pos;
    }

    /*
    if (indiv_id == 410)
      printf("Protein %d translate from %d to %d (%d)\n",protein_idx, c_pos,end_pos,protein_list[indiv_id][protein_idx]->leading_lagging);
    */
    int8_t value = 0;
    int8_t codon_list[64] = {};
    int8_t codon_idx = 0;
    int32_t count_loop = 0;

    bool contin = true;
    if (protein_list[indiv_id][protein_idx].leading_lagging == 0) {
      // LEADING

      while (count_loop<protein_list[indiv_id][protein_idx].protein_length/3 && codon_idx < 64) {
        value = 0;
        for (int8_t i = 0; i < 3; i++) {
          t_pos = c_pos + i >= dna_size[indiv_id] ? c_pos + i - dna_size[indiv_id] : c_pos + i;
          if (dna[indiv_id][t_pos] == '1' ) value += 1 << (CODON_SIZE - i - 1);
        }
        codon_list[codon_idx] = value;
        codon_idx++;

        count_loop++;
        c_pos+=3;
        c_pos = c_pos >= dna_size[indiv_id] ? c_pos - dna_size[indiv_id] : c_pos;
      }
    } else {
      // LAGGING
      while (count_loop<protein_list[indiv_id][protein_idx].protein_length/3 && codon_idx < 64) {
        value = 0;
        for (int8_t i = 0; i < 3; i++) {
          t_pos = c_pos - i < 0 ? dna_size[indiv_id] + (c_pos - i) : c_pos - i;
          if (dna[indiv_id][t_pos] != '1' ) value += 1 << (CODON_SIZE - i - 1);
        }
        codon_list[codon_idx] = value;
        codon_idx++;

        count_loop++;

        c_pos-=3;
        c_pos = c_pos < 0 ? c_pos + dna_size[indiv_id] : c_pos;
      }
    }

    double M = 0.0;
    double W = 0.0;
    double H = 0.0;

    int32_t nb_m = 0;
    int32_t nb_w = 0;
    int32_t nb_h = 0;

    bool bin_m = false; // Initializing to false will yield a conservation of the high weight bit
    bool bin_w = false; // when applying the XOR operator for the Gray to standard conversion
    bool bin_h = false;


    for (int i = 0; i < codon_idx; i++) {
      switch (codon_list[i])
      {
        case CODON_M0 :
        {
          // M codon found
          nb_m++;

          // Convert Gray code to "standard" binary code
          bin_m ^= false; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

          // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
          //~ M <<= 1;
          M *= 2;

          // Add this nucleotide's contribution to M
          if (bin_m) M += 1;

          break;
        }
        case CODON_M1 :
        {
          // M codon found
          nb_m++;

          // Convert Gray code to "standard" binary code
          bin_m ^= true; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

          // A lower-than-the-previous-lowest bit was found, make a left bitwise shift
          //~ M <<= 1;
          M *= 2;

          // Add this nucleotide's contribution to M
          if (bin_m) M += 1;

          break;
        }
        case CODON_W0 :
        {
          // W codon found
          nb_w++;

          // Convert Gray code to "standard" binary code
          bin_w ^= false; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

          // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
          //~ W <<= 1;
          W *= 2;

          // Add this nucleotide's contribution to W
          if (bin_w) W += 1;

          break;
        }
        case CODON_W1 :
        {
          // W codon found
          nb_w++;

          // Convert Gray code to "standard" binary code
          bin_w ^= true; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

          // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
          //~ W <<= 1;
          W *= 2;

          // Add this nucleotide's contribution to W
          if (bin_w) W += 1;

          break;
        }
        case CODON_H0 :
        case CODON_START : // Start codon codes for the same amino-acid as H0 codon
        {
          // H codon found
          nb_h++;

          // Convert Gray code to "standard" binary code
          bin_h ^= false; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

          // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
          //~ H <<= 1;
          H *= 2;

          // Add this nucleotide's contribution to H
          if (bin_h) H += 1;

          break;
        }
        case CODON_H1 :
        {
          // H codon found
          nb_h++;

          // Convert Gray code to "standard" binary code
          bin_h ^= true; // as bin_m was initialized to false, the XOR will have no effect on the high weight bit

          // A lower-than-the-previous-lowest weight bit was found, make a left bitwise shift
          //~ H <<= 1;
          H *= 2;

          // Add this nucleotide's contribution to H
          if (bin_h) H += 1;

          break;
        }
      }
    }



    //  ----------------------------------------------------------------------------------
    //  2) Normalize M, W and H values in [0;1] according to number of codons of each kind
    //  ----------------------------------------------------------------------------------
    protein_list[indiv_id][protein_idx].m = nb_m != 0 ? M / (pow(2, nb_m) - 1) : 0.5;
    protein_list[indiv_id][protein_idx].w = nb_w != 0 ? W / (pow(2, nb_w) - 1) : 0.0;
    protein_list[indiv_id][protein_idx].h = nb_h != 0 ? H / (pow(2, nb_h) - 1) : 0.5;

    //  ------------------------------------------------------------------------------------
    //  3) Normalize M, W and H values according to the allowed ranges (defined in macros.h)
    //  ------------------------------------------------------------------------------------
    // x_min <= M <= x_max
    // w_min <= W <= w_max
    // h_min <= H <= h_max
    protein_list[indiv_id][protein_idx].m  = (X_MAX - X_MIN) * protein_list[indiv_id][protein_idx].m + X_MIN;
    protein_list[indiv_id][protein_idx].w  = (w_max - W_MIN) * protein_list[indiv_id][protein_idx].w + W_MIN;
    protein_list[indiv_id][protein_idx].h  = (H_MAX - H_MIN) * protein_list[indiv_id][protein_idx].h + H_MIN;

    if ( nb_m == 0 || nb_w == 0 || nb_h == 0 || protein_list[indiv_id][protein_idx].w == 0.0 ||
        protein_list[indiv_id][protein_idx].h == 0.0 )
    {
      protein_list[indiv_id][protein_idx].is_functional = false;
    }
    else
    {
      protein_list[indiv_id][protein_idx].is_functional = true;
    }
  }
}


__global__ void compute_phenotype(int32_t* idx_protein, cProtein** protein_list,
                                  float** phenotype, int threads_size, int thread_dim) {
  int indiv_id = blockIdx.x / threads_size;
  int block_id = blockIdx.x % threads_size;
  int pos_block_size = threadIdx.x;

  int protein_idx = thread_dim*block_id+pos_block_size;

  if (protein_idx < idx_protein[indiv_id]) {
    if ( fabs(protein_list[indiv_id][protein_idx].w) < 1e-15 ||
        fabs(protein_list[indiv_id][protein_idx].h) < 1e-15 ) return;

    if (protein_list[indiv_id][protein_idx].is_functional) {

      // Compute triangle points' coordinates
      float x0 = protein_list[indiv_id][protein_idx].m -
                 protein_list[indiv_id][protein_idx].w;
      float x1 = protein_list[indiv_id][protein_idx].m;
      float x2 = protein_list[indiv_id][protein_idx].m +
                 protein_list[indiv_id][protein_idx].w;

      /*if (indiv_id == 991)
        printf("Protein %d : %f %f %f\n",protein_idx,protein_list[indiv_id][protein_idx]->m,
               protein_list[indiv_id][protein_idx]->w,protein_list[indiv_id][protein_idx]->h);*/

      int ix0 = (int) (x0 * 300);
      int ix1 = (int) (x1 * 300);
      int ix2 = (int) (x2 * 300);

      if (ix0 < 0) ix0 = 0; else if (ix0 > (299)) ix0 = 299;
      if (ix1 < 0) ix1 = 0; else if (ix1 > (299)) ix1 = 299;
      if (ix2 < 0) ix2 = 0; else if (ix2 > (299)) ix2 = 299;

      // Compute the first equation of the triangle
      float incY = (protein_list[indiv_id][protein_idx].h *
                    protein_list[indiv_id][protein_idx].e) / (ix1 - ix0);
      int count = 1;
      // Updating value between x0 and x1

      for (int i = ix0 + 1; i < ix1; i++) {
        atomicAdd(&(phenotype[indiv_id][i]), incY * (count++));
      }

      atomicAdd(&phenotype[indiv_id][ix1],
                (protein_list[indiv_id][protein_idx].h *
                 protein_list[indiv_id][protein_idx].e));

      // Compute the second equation of the triangle
      incY = (protein_list[indiv_id][protein_idx].h *
              protein_list[indiv_id][protein_idx].e) / (ix2 - ix1);
      count = 1;

      // Updating value between x1 and x2
      for (int i = ix1 + 1; i < ix2; i++) {
        atomicAdd(&phenotype[indiv_id][i],
                  ((protein_list[indiv_id][protein_idx].h *
                    protein_list[indiv_id][protein_idx].e) -
                   (incY * (count++))));
      }
    }
  }
}

__global__ void compute_metaerror_fitness(double selection_pressure,float** phenotype,
                                          float* target,
                                          float* metaerror, double* fitness) {
  int indiv_id = blockIdx.x;

  int fuzzy_idx = threadIdx.x;

  __shared__ float delta[300];

  if (phenotype[indiv_id][fuzzy_idx] > 1) phenotype[indiv_id][fuzzy_idx] = 1;
  if (phenotype[indiv_id][fuzzy_idx] < 0) phenotype[indiv_id][fuzzy_idx] = 0;

  delta[fuzzy_idx] = phenotype[indiv_id][fuzzy_idx] - target[fuzzy_idx];

  //if (indiv_id == 15) printf("DELTA[%d] = %f\n",fuzzy_idx,delta[fuzzy_idx]);

 /* if (threadIdx.x == 0) {
    metaerror[indiv_id] = 0;
  }*/

  __syncthreads();

  /*if (threadIdx.x < 299) {
    atomicAdd(metaerror+indiv_id,
              ((fabs(delta[fuzzy_idx]) +
                fabs(delta[fuzzy_idx + 1])) / (600.0)));
  }

  __syncthreads();*/

  if (threadIdx.x == 0) {
    metaerror[indiv_id] = 0;

    for (int i = 0; i < 299; i++) {
      metaerror[indiv_id] +=
                ((fabs(delta[i]) +
                  fabs(delta[i + 1])) / (600.0));
    }

    fitness[indiv_id] = exp(
        -selection_pressure * ((double)metaerror[indiv_id]));
  }
}

__global__ void free_list(cProtein** protein_list,
                          cRNA*** rna, int32_t* idx_protein,int32_t* idx_rna) {

  int indiv_id = blockIdx.x;

}

__global__ void debug_dna(size_t* dna_size, char** dna) {
  for (int i = 0; i < 1024; i++) {
    printf("DNA SIZE %d : %lu\n",i,dna_size[i]);

    for (size_t pos = 0; pos < dna_size[i]; pos++)
      dna[i][pos];
  }
}

__global__ void debug_promoter_start(size_t* dna_size,
                                     pStruct** dynPromoterList,
                                     int* nb_promoters, int indiv_id) {
  //int indiv_id = blockIdx.x;

 // for (int i = 0; i < 1024; i++)
 //   printf("DNA SIZE %d : %lu\n",i,dna_size[i]);
  printf("RNA %d : %d\n",indiv_id,nb_promoters[indiv_id]);

  printf("Individual %d (GPU) Promoters : LEADING ",indiv_id);
  // LEADING
  for (int idx = 0; idx < nb_promoters[indiv_id]; idx++) {
    if (dynPromoterList[indiv_id][idx].leading_or_lagging)
      printf("%d ",dynPromoterList[indiv_id][idx].pos);
  }
  printf("\n");

  printf("Individual %d (GPU) Promoters : LAGGING ",indiv_id);
  // LAGGING
  for (int idx = 0; idx < nb_promoters[indiv_id]; idx++) {
    if (!dynPromoterList[indiv_id][idx].leading_or_lagging)
      printf("%d ",dynPromoterList[indiv_id][idx].pos);
  }
  printf("END\n");
}


__global__ void debug_promoter_stop(size_t* dna_size,
                                     int8_t** dna_lead_term,
                                     int8_t** dna_lag_term, int* nb_promoters, int indiv_id) {
  //int indiv_id = blockIdx.x;

  // for (int i = 0; i < 1024; i++)
  //   printf("DNA SIZE %d : %lu\n",i,dna_size[i]);
  //printf("RNA %d : %d\n",indiv_id,nb_promoters[indiv_id]);

  printf("Term from CPU\n");
  printf("Individual %d (GPU) STOPs : LEADING ",indiv_id);
  // LEADING
  for (int pos = 0; pos < dna_size[indiv_id]; pos++) {
    if (dna_lead_term[indiv_id][pos] > 0) {
      printf("%d ",pos);
    }
  }
  printf("\n");

  printf("Individual %d (GPU) STOPs : LAGGING ",indiv_id);
  // LAGGING
  for (int pos = 0; pos < dna_size[indiv_id]; pos++) {
    if (dna_lag_term[indiv_id][pos] > 0) {
      printf("%d ",pos);
    }
  }
  printf("\n");
}

__global__ void debug_rna(size_t* dna_size,
                                     int8_t** dna_lead_term,
                                     int8_t** dna_lag_term,
                          cRNA*** rna,int32_t* idx_rna,
                          int indiv_id) {
  //int indiv_id = blockIdx.x;

  printf("Individual %d (GPU) %lu : \n",indiv_id,dna_size[indiv_id]);
  // LEADING
  for (int rna_idx = 0; rna_idx < idx_rna[indiv_id]; rna_idx++) {
    printf("RNA %d : ",rna_idx);
    if (rna[indiv_id][rna_idx]->leading_lagging == 0)
      printf("LEADING ");
    else
      printf("LAGGING ");

    printf("%d %d %f\n",rna[indiv_id][rna_idx]->begin,rna[indiv_id][rna_idx]->end,rna[indiv_id][rna_idx]->e);
  }
  printf("\n");
}

__global__ void debug_protein(int32_t* idx_protein,
                          cProtein** protein_list, char** dna,
                          int indiv_id) {
  printf("Individual %d (GPU) -- %d : \n",indiv_id,idx_protein[indiv_id]);

  for (int prot_idx = 0; prot_idx < idx_protein[indiv_id]; prot_idx++) {
    printf("Protein %d : %d %d %lf %lf %lf\n",prot_idx,
           protein_list[indiv_id][prot_idx].protein_start,
           protein_list[indiv_id][prot_idx].protein_end,
           protein_list[indiv_id][prot_idx].m,
           protein_list[indiv_id][prot_idx].h,
           protein_list[indiv_id][prot_idx].w);
  }

  printf("\n");

}

__global__ void debug_phenotype(float** phenotype,float* target, float* metaerror,
                                double* fitness,
                              int indiv_id) {
  printf("Individual %d (GPU) : \n", indiv_id);

  double delta_1[300];
  double metaerror_1 = 0;
  double fitness_1 = 0;

  for (int i = 0; i < 300; i++) {
    delta_1[i] = (double)phenotype[indiv_id][i] - (double)target[i];
  }

  for (int i = 0; i < 299; i++) {
    metaerror_1 +=
        (double)((fabs((double)delta_1[i]) +
          fabs((double)delta_1[i + 1])) / (double)(600.0));
  }

    //if (phenotype[indiv_id][i] != 0) {

  for (int i = 0; i < 300; i++) {

      if (phenotype[indiv_id][i] != 0) printf("[%d : %f]\n", i, phenotype[indiv_id][i]);


  }

  fitness_1 = exp(
      -1000.0 * (double)metaerror_1);

  printf("METAERROR %f --  %f // %e -- %e\n",metaerror_1,metaerror[indiv_id],
         fitness[indiv_id],fitness_1);
}

__global__ void debug_fitness(float** phenotype,float* target,
                              float* metaerror, double* fitness,
                                int indiv_id) {

  printf("Individual %d (GPU) : \n", indiv_id);


}
